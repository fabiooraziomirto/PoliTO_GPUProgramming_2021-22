
#include <hip/hip_runtime.h>
#include <stdio.h>


#define N 7
#define MASK_DIM 7

#define MASK_OFFSET (MASK_DIM/2)



__global__ void convolution_2(float *matrix, float *mask, float *result){

	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;


	int start_r = row - MASK_OFFSET;
	int start_c = col - MASK_OFFSET;

	float temp = 0;

	for(int i = 0; i < MASK_DIM; i++){
		for(int j = 0; j < MASK_DIM; j++){
			if((start_r + i) >= 0 && (start_r + i) < N){
				if((start_c + j) >= 0 && (start_c + j) < N){
					temp += matrix[(start_r+i) * N + start_c + j] * 
						mask[i * MASK_DIM + j];	
				}
			}
		}
	}
	
	result[row * N + col] = temp;
}

void generate_matrix(float *matrix, int dim){
	for(int i = 0; i < dim; i++){
		for(int j = 0; j < dim; j++){
			matrix[i * dim + j] = rand() % 20 + i*0.2;
		}
	}
}


void verify_result(float *m, float *mask, float *result){

	float temp;
	int off_r;
	int off_c;

	for(int i = 0; i < N; i++){
		for(int j = 0; j < N; j++){
			temp = 0;
		for(int k = 0; k < MASK_DIM; k++){
			off_r = i - MASK_OFFSET + k;
		for(int l = 0; l < N; l++){
			off_c = j - MASK_OFFSET + l;
		if(off_r >= 0 && off_r < N){
			if(off_c >= 0 && off_c < N){
				temp += m[off_r * N + off_c] * mask[k * MASK_DIM + l];
				}
			}
		}
	}
		result[i*N + j] = temp;
	}
	}
}


int main(void) {
	
	float *matrix;
	float *result;
	float *mask;

	matrix = (float *) malloc(N * N  * sizeof(float));
	
	result = (float *) malloc(N * N  * sizeof(float));

	mask = (float *) malloc(MASK_DIM * MASK_DIM  * sizeof(float));

	generate_matrix(matrix, N);
	generate_matrix(mask, MASK_DIM);

	for(int i = 0; i < N; i++){
		for(int j = 0; j < N; j++){
			printf("%f ", matrix[i * N +j]);
		}
		printf("\n");
	}


	
	float *matrix_dev, *mask_dev, *result_dev;
	
	
	hipMalloc(&matrix_dev, N * N * sizeof(float));
	hipMalloc(&mask_dev, MASK_DIM * MASK_DIM * sizeof(float));
	hipMalloc(&result_dev, N*N*sizeof(float));
	

	hipMemcpy(matrix_dev, matrix, N*N*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(mask_dev, mask, MASK_DIM * MASK_DIM *sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(result_dev, result, N*N*sizeof(float), hipMemcpyHostToDevice);
	
	int THREADS = 8;
	int BLOCKS = (N + THREADS - 1)/THREADS;

	dim3 gridDim(BLOCKS, BLOCKS);
	dim3 blockDim(THREADS, THREADS);

	convolution_2<<<gridDim, blockDim>>>(matrix_dev, mask_dev, result_dev);

	hipMemcpy(matrix, matrix_dev, N*N*sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy(result, result_dev, N*N*sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy(mask, mask_dev, MASK_DIM * MASK_DIM *sizeof(float), hipMemcpyDeviceToHost);
	

	printf("RESULT\n");
	for(int i = 0; i < N; i++){
		for(int j = 0; j < N; j++){
			printf("%f ", result[i * N +j]);
		}
		printf("\n");
	}

	verify_result(matrix, mask, result);

	printf("RESULT\n");
	for(int i = 0; i < N; i++){
		for(int j = 0; j < N; j++){
			printf("%f ", result[i * N +j]);
		}
		printf("\n");
	}

	free(result);
	free(matrix);

	hipFree(matrix_dev);
	hipFree(mask_dev);
	hipFree(result_dev);


	return 0;
}
