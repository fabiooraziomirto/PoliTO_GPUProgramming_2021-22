
#include <hip/hip_runtime.h>
#include <stdio.h>

#define NL 2048
#define ML 8

__constant__ float CM[8] = {3, 4, 5, 6, 6, 5, 4, 3};

__global__ void convolution_1(float *M, float *P, float *N, int Mask_Width, int vector_Width){

	int tid= blockIdx.x * blockDim.x + threadIdx.x;
	float Pvalue = 0;
	int N_start_point = tid - (Mask_Width/2);

	for (int j = 0; j < Mask_Width; j++){
		if (N_start_point + j >= 0 && N_start_point+ j < vector_Width){
			Pvalue += N[N_start_point+ j] * CM[j];
		}
	}
	P[tid] = Pvalue;
}

int main(void) {
	
	int j;
	float *P, *N;
	float *dev_P,*dev_N;
	
	//M = (float *) malloc(ML * sizeof(float));
	P = (float *) malloc(NL * sizeof(float));
	N = (float *) malloc(NL * sizeof(float));

	//cudaMalloc(&dev_M, ML * sizeof(float));
	hipMalloc(&dev_P, NL * sizeof(float));
	hipMalloc(&dev_N, NL * sizeof(float));

	for (j = 0; j < NL; j++) {
		N[j] = j;
	}

	
	// run version with static shared memory
	
	//cudaMemcpy(dev_M, M, ML*sizeof(float), cudaMemcpyHostToDevice);
	hipMemcpy(dev_N, N, NL*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dev_P, P, NL*sizeof(float), hipMemcpyHostToDevice);

	convolution_1<<<16, 128>>>(CM, dev_P, dev_N, ML, NL);
	// work also 
	// convolution_1<<<8, 256>>>(dev_M, dev_P, dev_N, ML, NL);
	//cudaMemcpy(M, dev_M, ML*sizeof(float), cudaMemcpyDeviceToHost);
	hipMemcpy(N, dev_N, NL*sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy(P, dev_P, NL*sizeof(float), hipMemcpyDeviceToHost);

	printf("P = ");
	for (j = 0; j < NL; j++) {
		printf("%f ", P[j]);
	}	

	printf("\n");

	//free(M);
	free(P);
	free(N);

	//cudaFree(dev_M);
	hipFree(dev_P);
	hipFree(dev_N);

	return 0;
}
