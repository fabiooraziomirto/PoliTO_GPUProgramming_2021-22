
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>

#define N 3

__global__ void simple_kernel(float* x, float* y) {

	int i = threadIdx.x + blockDim.x * blockIdx.x;
	if ( (i % blockDim.x) == 0)
		x[i] = x[i] + 1;
	if ( (i % blockDim.x) == 1)
		y[i] = x[i-1] / 2.0;
	if ( (i % blockDim.x) == 2)
		x[i-1] = cos(y[i-1]);
	if ( (i % blockDim.x) == 3)
		y[i-1] = x[i-2] * y[i-2] + x[i-3];
	__syncthreads();

}

int main(void) {

	float *x, *y;
	float *dev_x, *dev_y;
	
	x = (float *) malloc(N * sizeof(float));
	y = (float *) malloc(N * sizeof(float));

	hipMalloc(&dev_x, N * sizeof(float));
	hipMalloc(&dev_y, N * sizeof(float));
	
	x[0] = 45;

	// run version with static shared memory
	hipMemcpy(dev_x, x, N*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dev_y, y, N*sizeof(float), hipMemcpyHostToDevice);

	simple_kernel<<<1, 32>>>(dev_x, dev_y);

	hipMemcpy(x, dev_x, N*sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy(y, dev_y, N*sizeof(float), hipMemcpyDeviceToHost);

	for (int i = 0; i < N; i++) {
		printf("x = %f ", x[i]);
		printf("y = %f ", y[i]);
	}
	printf("\n");

	free(x);
	free(y);

	hipFree(dev_x);
	hipFree(dev_y);

	return 0;
}
